#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/cuda/saturate_cast.hpp>

using namespace cv;

namespace OVR
{
	namespace CUDA
	{
		__global__ void remap_kernel(const cuda::PtrStepSz<uchar3> src, const cuda::PtrStep<float> mapx, const cuda::PtrStep<float> mapy, cuda::PtrStepSz<uchar3> dst)
		{
			const int x = blockDim.x * blockIdx.x + threadIdx.x;
			const int y = blockDim.y * blockIdx.y + threadIdx.y;

			if (x < dst.cols && y < dst.rows)
			{
				float xcoo = mapx.ptr(y)[x];
				float ycoo = mapy.ptr(y)[x];
				int X = trunc(xcoo);
				int Y = trunc(ycoo);
				float xfrac = xcoo - X;
				float yfrac = ycoo - Y;
				if (0 <= X && X < src.cols && 0 <= Y && Y < src.rows)
				{
					//uchar3 p[2][2];
					uchar3 p00 = src(Y, X);
					uchar3 p10 = src(Y + 1, X);
					uchar3 p01 = src(Y, X + 1);
					uchar3 p11 = src(Y + 1, X + 1);
					// bilinear interpolation 
					dst.ptr(y)[x].x = (p00.x * (1 - xfrac) + p01.x * xfrac) * (1 - yfrac) + (p10.x * (1 - xfrac) + p11.x * xfrac) * yfrac;
					dst.ptr(y)[x].y = (p00.y * (1 - xfrac) + p01.y * xfrac) * (1 - yfrac) + (p10.y * (1 - xfrac) + p11.y * xfrac) * yfrac;
					dst.ptr(y)[x].z = (p00.z * (1 - xfrac) + p01.z * xfrac) * (1 - yfrac) + (p10.z * (1 - xfrac) + p11.z * xfrac) * yfrac;
				}
			}
		}

		double remap(const cuda::GpuMat src, cuda::GpuMat dst, const cuda::GpuMat mapx, const cuda::GpuMat mapy)
		{
			dim3 threads(16, 16);
			dim3 grid((src.cols) / (threads.x), (src.rows) / (threads.y));

			int64 start = getTickCount();
			remap_kernel<<<grid, threads>>>(src, mapx, mapy, dst);
			int64 stop = getTickCount();

			hipDeviceSynchronize();
			return (stop - start) * 1000000 / getTickFrequency();
		}
	}
}