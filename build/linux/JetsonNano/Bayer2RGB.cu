#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgproc/imgproc.hpp>


using namespace cv;

namespace OVR
{
	namespace CUDA
	{
		struct Bayer2BGR
		{
			ushort patch[4][4];
			__device__ void shr_8()
			{
				patch[0][0] >>= 8;
				patch[0][1] >>= 8;
				patch[0][2] >>= 8;
				patch[0][3] >>= 8;
				patch[1][0] >>= 8;
				patch[1][1] >>= 8;
				patch[1][2] >>= 8;
				patch[1][3] >>= 8;
				patch[2][0] >>= 8;
				patch[2][1] >>= 8;
				patch[2][2] >>= 8;
				patch[2][3] >>= 8;
				patch[3][0] >>= 8;
				patch[3][1] >>= 8;
				patch[3][2] >>= 8;
				patch[3][3] >>= 8;
			}

			__device__ void apply(int x, int y, cuda::PtrStepSz<uchar3> dst)
			{
				uchar3 bgr[2][2];
				ushort p[4][4];
				p[0][0] = patch[0][0] & 0xff;
				p[0][1] = patch[0][1] & 0xff;
				p[0][2] = patch[0][2] & 0xff;
				p[0][3] = patch[0][3] & 0xff;
				p[1][0] = patch[1][0] & 0xff;
				p[1][1] = patch[1][1] & 0xff;
				p[1][2] = patch[1][2] & 0xff;
				p[1][3] = patch[1][3] & 0xff;
				p[2][0] = patch[2][0] & 0xff;
				p[2][1] = patch[2][1] & 0xff;
				p[2][2] = patch[2][2] & 0xff;
				p[2][3] = patch[2][3] & 0xff;
				p[3][0] = patch[3][0] & 0xff;
				p[3][1] = patch[3][1] & 0xff;
				p[3][2] = patch[3][2] & 0xff;
				p[3][3] = patch[3][3] & 0xff;

				bgr[0][0].x = (uchar)((p[0][1] + p[2][1]) >> 1); // B 
				bgr[0][0].y = (uchar)(p[1][1]); // G
				bgr[0][0].z = (uchar)((p[1][0] + p[1][2]) >> 1); // R

				bgr[0][1].x = (uchar)(p[2][1]); // B
				bgr[0][1].y = (uchar)((p[2][0] + p[2][2] + p[1][1] + p[3][1]) >> 2); // G
				bgr[0][1].z = (uchar)((p[1][0] + p[3][0] + p[1][2] + p[3][2]) >> 2); // R

				bgr[1][0].x = (uchar)((p[0][1] + p[2][1] + p[0][3] + p[2][3]) >> 2); // B
				bgr[1][0].y = (uchar)((p[0][2] + p[2][2] + p[1][1] + p[1][3]) >> 2); // G
				bgr[1][0].z = (uchar)(p[1][2]); // R
					
				bgr[1][1].x = (uchar)((p[2][1] + p[2][3]) >> 1); // B 
				bgr[1][1].y = (uchar)(p[2][2]); // G
				bgr[1][1].z = (uchar)((p[1][2] + p[3][2]) >> 1); // R

				// store result
				((uchar3 *)dst.ptr(y))[x] = bgr[0][0];
				((uchar3 *)dst.ptr(y))[x + 1] = bgr[1][0];
				((uchar3 *)dst.ptr(y + 1))[x] = bgr[0][1];
				((uchar3 *)dst.ptr(y + 1))[x + 1] = bgr[1][1];
			}

			// First row of image
			__device__ void applyUpper(int x, int y, cuda::PtrStepSz<uchar3> dst)
			{
				uchar3 bgr[2][2];
				ushort p[4][4];
				p[0][0] = patch[0][0] & 0xff;
				p[0][1] = patch[0][1] & 0xff;
				p[0][2] = patch[0][2] & 0xff;
				p[0][3] = patch[0][3] & 0xff;
				p[1][0] = patch[1][0] & 0xff;
				p[1][1] = patch[1][1] & 0xff;
				p[1][2] = patch[1][2] & 0xff;
				p[1][3] = patch[1][3] & 0xff;
				p[2][0] = patch[2][0] & 0xff;
				p[2][1] = patch[2][1] & 0xff;
				p[2][2] = patch[2][2] & 0xff;
				p[2][3] = patch[2][3] & 0xff;
				p[3][0] = patch[3][0] & 0xff;
				p[3][1] = patch[3][1] & 0xff;
				p[3][2] = patch[3][2] & 0xff;
				p[3][3] = patch[3][3] & 0xff;

				bgr[0][0].x = (uchar)((p[0][1] + p[2][1]) >> 1); // B 
				bgr[0][0].y = (uchar)(p[1][1]); // G
				bgr[0][0].z = (uchar)((p[1][0] + p[1][2]) >> 1); // R

				bgr[0][1].x = (uchar)(p[2][1]); // B
				bgr[0][1].y = (uchar)((p[2][0] + p[2][2] + p[1][1] + p[3][1]) >> 2); // G
				bgr[0][1].z = (uchar)((p[1][0] + p[3][0] + p[1][2] + p[3][2]) >> 2); // R

				bgr[1][0].x = (uchar)((p[0][1] + p[2][1] + p[0][3] + p[2][3]) >> 2); // B
				bgr[1][0].y = (uchar)((p[0][2] + p[2][2] + p[1][1] + p[1][3]) >> 2); // G
				bgr[1][0].z = (uchar)(p[1][2]); // R
					
				bgr[1][1].x = (uchar)((p[2][1] + p[2][3]) >> 1); // B 
				bgr[1][1].y = (uchar)(p[2][2]); // G
				bgr[1][1].z = (uchar)((p[1][2] + p[3][2]) >> 1); // R

				// store result
				((uchar3 *)dst.ptr(y))[x] = bgr[0][0];
				((uchar3 *)dst.ptr(y))[x + 1] = bgr[1][0];
				((uchar3 *)dst.ptr(y + 1))[x] = bgr[0][1];
				((uchar3 *)dst.ptr(y + 1))[x + 1] = bgr[1][1];
			}
		};

		// Stereo bayer to RGB conversion
		// each thread calculates pixels of interest
		//
		// G|RG|R
		// ------
		// B|GB|G GB -- pixels of interest
		// G|RG|R RG -- pixels of interest
		// ------
		// B|GB|G
		// 
		__global__ void bayer2BGR(const cuda::PtrStepSz<ushort> src, cuda::PtrStepSz<uchar3> left, cuda::PtrStepSz<uchar3> right)
		{
			int s_x = 2 * ((blockIdx.x * blockDim.x) + threadIdx.x);
			int s_y = 2 * ((blockIdx.y * blockDim.y) + threadIdx.y);

			// pixels of interest
			Bayer2BGR bayer;

			if (0 < s_y && s_y < src.rows - 2 && 0 < s_x && s_x < src.cols - 2)
			{
				bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
				//
				bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
				//
				bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				//
				bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else if (0 == s_y) // first row
			{
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				//
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				//
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				//
				if (s_x == 0)
				{
					bayer.patch[1][0] = bayer.patch[1][2];
					bayer.patch[2][0] = bayer.patch[2][2];
					bayer.patch[3][0] = bayer.patch[3][2];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
					bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];
				} 
				else if (s_x == src.cols)
				{
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
					bayer.patch[1][3] = bayer.patch[1][1];
					bayer.patch[2][3] = bayer.patch[2][1];
					bayer.patch[3][3] = bayer.patch[3][1];
				}
				else 
				{
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
					bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];
				}

				bayer.patch[0][0] = bayer.patch[2][0];
				bayer.patch[0][1] = bayer.patch[2][1];
				bayer.patch[0][2] = bayer.patch[2][2];
				bayer.patch[0][3] = bayer.patch[2][3];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else if (s_y == src.rows - 2) // last row
			{
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				//
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				//
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				//
				if (s_x == 0)
				{
					bayer.patch[0][0] = bayer.patch[0][2];
					bayer.patch[1][0] = bayer.patch[1][2];
					bayer.patch[2][0] = bayer.patch[2][2];
					bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				} 
				else if (s_x == src.cols)
				{
					bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[0][3] = bayer.patch[0][1];
					bayer.patch[1][3] = bayer.patch[1][1];
					bayer.patch[2][3] = bayer.patch[2][1];
				}
				else 
				{
					bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				}

				bayer.patch[3][0] = bayer.patch[1][0];
				bayer.patch[3][1] = bayer.patch[1][1];
				bayer.patch[3][2] = bayer.patch[1][2];
				bayer.patch[3][3] = bayer.patch[1][3];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else if (0 == s_x) // first col
			{
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
				//
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
				//
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				//
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];

				bayer.patch[0][0] = bayer.patch[0][2];
				bayer.patch[1][0] = bayer.patch[1][2];
				bayer.patch[2][0] = bayer.patch[2][2];
				bayer.patch[3][0] = bayer.patch[3][2];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else // last col
			{
				bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				//
				bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				//
				bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				//
				bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				//
				bayer.patch[0][3] = bayer.patch[0][1];
				bayer.patch[1][3] = bayer.patch[1][1];
				bayer.patch[2][3] = bayer.patch[2][1];
				bayer.patch[3][3] = bayer.patch[3][1];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
		}

		double bayerGB2BGR(cuda::GpuMat src, cuda::GpuMat left, cuda::GpuMat right)
		{
			dim3 threads(16, 16);
			dim3 grid((src.cols / 2) / (threads.x), (src.rows / 2) / (threads.y));
			hipDeviceSynchronize();
			int64 start = getTickCount();
			bayer2BGR<<<grid, threads>>>(src, left, right);
			int64 stop = getTickCount();
			hipDeviceSynchronize();
			return (stop - start) * 1000000 / getTickFrequency();
			//return 0;
		}
	}
}
